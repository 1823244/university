#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 1
#define MAX_BLOCKS 65535

/* максимальное количество потоков в блоке;
 * получено так:
 * cudaGetDeviceProperties(&deviceProp, i);
 * printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock); 
 */
#define MAX_THREADS 512

__device__ __host__ float f(float x)
{
	return sqrtf(x - 1) + 1 / (x - 3);
}

__global__ void kernel(long long countStep, float h, float *result)
{
	long long step = blockIdx.y * MAX_BLOCKS * MAX_THREADS + (blockIdx.x * blockDim.x + threadIdx.x);
    
	if (step > countStep) {
		return;
    }

	result[step] = f(1 + h * step);
}

__host__ void calculateOnHost(long long countStep, float h, float *result) 
{
	for (long long i = 0; i < countStep; i++)
	{
		result[i] = f(1 + h * i);
	}
}

int main(int argc, char** argv)
{
    int multiplier = 4;
    printf("Step: 1 * 10^%d\n", multiplier);

    float k = N * powf(10.0, multiplier);
    float h = N / k;
    printf("k = %f, h = %f\n", k, h);

	// вычисляем количество шагов
    long long countStep = N / h + 1;

	// подсчитываем количество блоков
    long long countBlocks = countStep / MAX_THREADS;

	// если после деления остался остаток -- добавляем ещё блок
    if (countStep % MAX_THREADS != 0) {
        countBlocks++;
    }

	int countY = 1;
	if (countBlocks > MAX_BLOCKS) {
		countY = 1 + countBlocks / MAX_BLOCKS;
		countBlocks = MAX_BLOCKS;
	}
  
	// высчитываем количество блоков и строк в сетке
	// def: dim3(unsigned int vx = 1, unsigned int vy = 1, unsigned int vz = 1
    dim3 gridDim = dim3(countBlocks, countY);
    dim3 blockDim = dim3(MAX_THREADS);
    
    float *result = new float[countStep];
    float *resultDev;
    long long size = countStep * sizeof(float);

	printf("countStep = %d\n", countStep);
	printf("countBlocks = %d\n", countBlocks);

	/* CUDA START */
	// del
	time_t startTimeHost = clock();
	result = new float[countStep]; 
	// end del

	// инициализируем события, чтобы потом посчитать время
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// запускаем событие начала
	hipEventRecord(start, 0);
	// выделяем память
	hipMalloc((void**) &resultDev, size);
	printf("Start CUDA from %d to %d with step = %0.8f\n", 1, N + 1, h);
	// запускаем вычисления
	kernel<<<gridDim, blockDim>>>(countStep, h, resultDev);

	// барьерная синхронизация
	hipDeviceSynchronize();
	// копируем данные в result из resultDev размера size из GPU
	hipMemcpy(result, resultDev, size, hipMemcpyDeviceToHost);
	// чистим память
	hipFree(resultDev);
	
	// запускаем событие завершения работы
	hipEventRecord(stop, 0);
	// ждём, пока всё завершится
	hipEventSynchronize(stop);
	printf("Stop CUDA\n");

	float totalTimeDevice;
	// вычисляем время
	hipEventElapsedTime(&totalTimeDevice, start, stop);

	// del
	time_t endTimeHost = clock();
	double totalTimeHost = (double)(endTimeHost - startTimeHost) / CLOCKS_PER_SEC * 1000;
	// end del
	printf("\nTime on device (ms): %1.3lf\n", totalTimeHost);
	
	// удаляем события
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// чистим память
	free(result);
	result = NULL;

	/* CUDA END */

	/* HOST START */

	// засекаем время
	startTimeHost = clock();

	result = new float[countStep]; 

	// подсчитываем
	calculateOnHost(countStep, h, result);

	// считаем разницу во времени
	endTimeHost = clock();
	totalTimeHost = (double)(endTimeHost - startTimeHost) / CLOCKS_PER_SEC * 1000;

	printf("\nTime on host (ms): %1.3lf\n", totalTimeHost);

	/* HOST END */

	getchar();

	// чистим память
	free(result);
	hipDeviceReset();

	return 0;
}
