#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <climits>
#include <hip/hip_runtime.h>

#define MAX_BLOCKS 65535

/* максимальное количество потоков в блоке;
 * получено так:
 * cudaGetDeviceProperties(&deviceProp, i);
 * printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock); 
 */
#define MAX_THREADS 512

// количество итераций в каждом потоке
#define PARTS 1000

__global__ void kernel(long long countStep, int *vector, int *result)
{
	long long step = blockIdx.y * MAX_BLOCKS * MAX_THREADS + (blockIdx.x * blockDim.x + threadIdx.x);
    
	if (step > countStep) {
		return;
    }
	
	// находим минимальное
	int min = INT_MAX;
	for (long long i = step * PARTS; i < step + PARTS; i++)
	{
		if (min > vector[i]) {
			min = vector[i];
		}
	}

	result[step] = min;
}

__host__ int calculateOnHost(long long countStep, int *vector) 
{
	// находим минимальное
	int min = INT_MAX;
	for (long long i = 0; i < countStep; i++)
	{
		if (min > vector[i]) {
			min = vector[i];
		}
	}

	return min;
}

int main(int argc, char** argv)
{
    int multiplier = 6;
    printf("Array size: 10^%d\n", multiplier);

    long long countStep = (long long)pow(10.0, multiplier);

	// объявлем массив и заполняем его данными
	int *vector = new int[countStep];
	srand(time(NULL));
	for (long long i = 0; i < countStep; i++)
	{
		vector[i] = rand();
	}

	// подсчитываем количество блоков
    long long countBlocks = countStep / PARTS;

	// если после деления остался остаток -- добавляем ещё блок
    if (countStep % PARTS != 0) {
        countBlocks++;
    }

	// высчитываем количество блоков и строк в сетке
	int countY = 1;
	if (countBlocks > MAX_BLOCKS) {
		countY = 1 + countBlocks / MAX_BLOCKS;
		countBlocks = MAX_BLOCKS;
	}
	
	// def: dim3(unsigned int vx = 1, unsigned int vy = 1, unsigned int vz = 1
    dim3 gridDim = dim3(countBlocks, countY);
    dim3 blockDim = dim3(PARTS);
    
    int *result = new int[countStep];
    long long size = countStep * sizeof(int);

	printf("countStep = %d\n", countStep);

	/* CUDA START */
	// del
	time_t startTimeHost = clock();
	// end del

	// инициализируем события, чтобы потом посчитать время
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// запускаем событие начала
	hipEventRecord(start, 0);
	
	printf("Start CUDA\n");

	int count = countStep;
	while (count) {
		int *resultDev;
		// выделяем память
		long long size = count * sizeof(int);
		hipMalloc((void**) &resultDev, size);
		// запускаем вычисления
		kernel<<<gridDim, blockDim>>>(count, vector, resultDev);
		// барьерная синхронизация
		hipDeviceSynchronize();
		// копируем данные в vector из resultDev размера size из GPU
		hipMemcpy(vector, resultDev, size, hipMemcpyDeviceToHost);
		// чистим память
		hipFree(resultDev);
		count /= PARTS;
	}
	
	// запускаем событие завершения работы
	hipEventRecord(stop, 0);
	// ждём, пока всё завершится
	hipEventSynchronize(stop);
	printf("Stop CUDA\n");

	float totalTimeDevice;
	// вычисляем время
	hipEventElapsedTime(&totalTimeDevice, start, stop);

	// del
	time_t endTimeHost = clock();
	double totalTimeHost = (double)(endTimeHost - startTimeHost) / CLOCKS_PER_SEC * 1000;
	// end del
	printf("\nTime on device (ms): %1.3lf\n", totalTimeHost);
	
	// удаляем события
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// чистим память
	free(vector);
	vector = NULL;

	/* CUDA END */

	/* HOST START */
	vector = new int[countStep];
	for (long long i = 0; i < countStep; i++)
	{
		vector[i] = rand();
	}

	// засекаем время
	startTimeHost = clock();

	// подсчитываем
	calculateOnHost(countStep, vector);

	// считаем разницу во времени
	endTimeHost = clock();
	totalTimeHost = (double)(endTimeHost - startTimeHost) / CLOCKS_PER_SEC * 1000;

	printf("\nTime on host (ms): %1.3lf\n", totalTimeHost);

	/* HOST END */

	getchar();

	// чистим память
	free(vector);
	hipDeviceReset();

	return 0;
}
